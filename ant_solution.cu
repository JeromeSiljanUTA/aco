#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>

#define NUM_ANTS 8
#define NUM_NODES 8

__global__ void ant_solution_kernel(float *distances_matrix,
                                    float *pheromones_matrix,
                                    int *prev_visited_matrix) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= NUM_ANTS)
    return;

  for (int i = 0; i < NUM_NODES; i++) {
    printf("Index: %0.2d\tvalue: %0.2f\n", idx * NUM_NODES + i,
           pheromones_matrix[idx * NUM_NODES + i]);
  }
}

extern "C" {
void ant_solution(float *distances_matrix, float *pheromones_matrix,
                  int *prev_visited_matrix) {

  float *d_distances_matrix, *d_pheromones_matrix;
  int *d_prev_visited_matrix;

  hipMalloc((void **)&d_distances_matrix,
             NUM_ANTS * NUM_NODES * sizeof(float));
  hipMalloc((void **)&d_pheromones_matrix,
             NUM_ANTS * NUM_NODES * sizeof(float));
  hipMalloc((void **)&d_prev_visited_matrix,
             NUM_ANTS * NUM_NODES * sizeof(int));

  hipMemcpy(d_distances_matrix, distances_matrix,
             NUM_ANTS * NUM_NODES * sizeof(float), hipMemcpyHostToDevice);

  hipMemcpy(d_pheromones_matrix, pheromones_matrix,
             NUM_ANTS * NUM_NODES * sizeof(float), hipMemcpyHostToDevice);

  hipMemcpy(d_prev_visited_matrix, prev_visited_matrix,
             NUM_ANTS * NUM_NODES * sizeof(int), hipMemcpyHostToDevice);

  ant_solution_kernel<<<NUM_ANTS, 1>>>(d_distances_matrix, d_pheromones_matrix,
                                       d_prev_visited_matrix);

  hipFree(d_prev_visited_matrix);
}
}