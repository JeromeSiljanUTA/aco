#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>

#define NUM_ANTS 8

__global__ void dist_sum_kernel(float *prev_visited, float *distances,
                                float *dist_sums) {
  int num_nodes = NUM_ANTS;
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= num_nodes) {
    return;
  }
  for (int i = 0; i < num_nodes - 1; i++) {
    int current_node = (int)prev_visited[i];
    int next_node = (int)prev_visited[i + 1];
    dist_sums[idx] += distances[(NUM_ANTS * next_node) + current_node];
  }
  dist_sums[idx] +=
      distances[NUM_ANTS * (int)prev_visited[0] + (int)prev_visited[num_nodes]];
}

extern "C" {
void dist_sum(float *prev_visited, float *distances, float *dist_sums) {

  size_t size = (size_t)(NUM_ANTS * NUM_ANTS);
  float *d_prev_visited, *d_distances, *d_dist_sums;

  // Assuming one ant per node
  hipMalloc((void **)&d_prev_visited, size * sizeof(float));
  hipMalloc((void **)&d_distances, size * sizeof(float));
  hipMalloc((void **)&d_dist_sums, NUM_ANTS * sizeof(float));

  hipMemcpy(d_prev_visited, prev_visited, size * sizeof(float),
             hipMemcpyHostToDevice);
  hipMemcpy(d_distances, distances, size * sizeof(float),
             hipMemcpyHostToDevice);

  dist_sum_kernel<<<NUM_ANTS, 1>>>(d_prev_visited, d_distances, d_dist_sums);

  hipMemcpy(dist_sums, d_dist_sums, NUM_ANTS * sizeof(float),
             hipMemcpyDeviceToHost);

  hipFree(d_prev_visited);
  hipFree(d_distances);
  hipFree(d_dist_sums);
}
}